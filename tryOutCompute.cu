#include "hip/hip_runtime.h"
#include "bitonic_sort.cuh"
#include "gpus/gpu_csr_kernel.h"

__inline__ __device__
void scan_128(volatile short *s_scan)
{
    int ai, bi;
    int baseai = 1 + 2 * threadIdx.x;
    int basebi = baseai + 1;
    short temp;

    if (threadIdx.x < 64) { ai = baseai - 1;     bi = basebi - 1;     s_scan[bi] += s_scan[ai]; }
    __syncthreads();
    if (threadIdx.x < 32) { ai =  2 * baseai - 1;  bi =  2 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    __syncthreads();
    if (threadIdx.x < 16) { ai =  4 * baseai - 1;  bi =  4 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x < 8)  { ai = 8 * baseai - 1;  bi = 8 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x < 4)  { ai = 16 * baseai - 1;  bi = 16 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x < 2)  { ai = 32 * baseai - 1;  bi = 32 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x == 0) { s_scan[127] += s_scan[63]; s_scan[128] = s_scan[127]; s_scan[127] = 0; temp = s_scan[63]; s_scan[63] = 0; s_scan[127] += temp; }
    if (threadIdx.x < 2)  { ai = 32 * baseai - 1;  bi = 32 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 4)  { ai = 16 * baseai - 1;  bi = 16 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 8)  { ai = 8 * baseai - 1;  bi = 8 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 16) { ai =  4 * baseai - 1;  bi =  4 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 32) { ai =  2 * baseai - 1;  bi =  2 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    __syncthreads();
    if (threadIdx.x < 64) { ai = baseai - 1;   bi = basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp; }
}

__inline__ __device__
void scan_512(volatile short *s_scan)
{
    int ai, bi;
    int baseai = 1 + 2 * threadIdx.x;
    int basebi = baseai + 1;
    short temp;

    if (threadIdx.x < 256) { ai = baseai - 1;     bi = basebi - 1;     s_scan[bi] += s_scan[ai]; }
    __syncthreads();
    if (threadIdx.x < 128) { ai =  2 * baseai - 1;  bi =  2 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    __syncthreads();
    if (threadIdx.x < 64)  { ai =  4 * baseai - 1;  bi =  4 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    __syncthreads();
    if (threadIdx.x < 32) { ai =  8 * baseai - 1;  bi =  8 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    __syncthreads();
    if (threadIdx.x < 16) { ai =  16 * baseai - 1;  bi =  16 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x < 8)  { ai = 32 * baseai - 1;  bi = 32 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x < 4)  { ai = 64 * baseai - 1;  bi = 64 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x < 2)  { ai = 128 * baseai - 1;  bi = 128 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x == 0) { s_scan[511] += s_scan[255]; s_scan[512] = s_scan[511]; s_scan[511] = 0; temp = s_scan[255]; s_scan[255] = 0; s_scan[511] += temp; }
    if (threadIdx.x < 2)  { ai = 128 * baseai - 1;  bi = 128 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 4)  { ai = 64 * baseai - 1;  bi = 64 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 8)  { ai = 32 * baseai - 1;  bi = 32 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 16) { ai =  16 * baseai - 1;  bi =  16 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 32) { ai =  8 * baseai - 1;  bi =  8 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    __syncthreads();
    if (threadIdx.x < 64) { ai =  4 * baseai - 1;  bi =  4 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    __syncthreads();
    if (threadIdx.x < 128) { ai =  2 * baseai - 1;  bi =  2 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    __syncthreads();
    if (threadIdx.x < 256) { ai = baseai - 1;   bi = basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp; }
}

__inline__ __device__
void scan_256(volatile short *s_scan)
{
    int ai, bi;
    int baseai = 1 + 2 * threadIdx.x;
    int basebi = baseai + 1;
    short temp;

    if (threadIdx.x < 128) { ai = baseai - 1;     bi = basebi - 1;     s_scan[bi] += s_scan[ai]; }
    __syncthreads();
    if (threadIdx.x < 64) { ai =  2 * baseai - 1;  bi =  2 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    __syncthreads();
    if (threadIdx.x < 32) { ai =  4 * baseai - 1;  bi =  4 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    __syncthreads();
    if (threadIdx.x < 16) { ai =  8 * baseai - 1;  bi =  8 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x < 8)  { ai = 16 * baseai - 1;  bi = 16 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x < 4)  { ai = 32 * baseai - 1;  bi = 32 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x < 2)  { ai = 64 * baseai - 1;  bi = 64 * basebi - 1;   s_scan[bi] += s_scan[ai]; }
    if (threadIdx.x == 0) { s_scan[255] += s_scan[127]; s_scan[256] = s_scan[255]; s_scan[255] = 0; temp = s_scan[127]; s_scan[127] = 0; s_scan[255] += temp; }
    if (threadIdx.x < 2)  { ai = 64 * baseai - 1;  bi = 64 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 4)  { ai = 32 * baseai - 1;  bi = 32 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 8)  { ai = 16 * baseai - 1;  bi = 16 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 16) { ai =  8 * baseai - 1;  bi =  8 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    if (threadIdx.x < 32) { ai =  4 * baseai - 1;  bi =  4 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    __syncthreads();
    if (threadIdx.x < 64) { ai =  2 * baseai - 1;  bi =  2 * basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp;}
    __syncthreads();
    if (threadIdx.x < 128) { ai = baseai - 1;   bi = basebi - 1;   temp = s_scan[ai]; s_scan[ai] = s_scan[bi]; s_scan[bi] += temp; }
}

/*
template<typename sT, typename T>
__inline__ __device__
void scan_double_width_plus1_shfl(volatile  sT *s_scan,
                                  volatile  T *s_scan_shfl,
                                  const int     local_id,
                                  T r_in,
                                  T r_in_halfwidth,
                                  const int seg_num)
{
    // 3-stage method. scan-scan-propogate

    // shfl version
    const int lane_id = local_id % 32;
    const int seg_id = local_id / 32;

    // stage 1. thread bunch scan
    T r_scan = scan_32_shfl<T>(r_in, lane_id);
    T r_scan_halfwidth = scan_32_shfl<T>(r_in_halfwidth, lane_id);

    if (lane_id == 32 - 1)
    {
        s_scan_shfl[seg_id] = r_scan;
        s_scan_shfl[seg_id + seg_num] = r_scan_halfwidth;
    }

    // inclusive to exclusive
    r_scan = __shfl_up(r_scan, 1);
    r_scan_halfwidth = __shfl_up(r_scan_halfwidth, 1);
    r_scan = lane_id ? r_scan : 0;
    r_scan_halfwidth = lane_id ? r_scan_halfwidth : 0;

    __syncthreads();

    // stage 2. one thread bunch scan
    r_in = (local_id < 2 * seg_num) ? s_scan_shfl[local_id] : 0;
    if (!seg_id)
        r_in = scan_32_shfl<T>(r_in, lane_id);

    if (local_id < 2 * seg_num)
        s_scan_shfl[local_id + 1] = r_in;

    // single thread in-place scan
    //scan_single<T>(s_scan_shfl, local_id, seg_num+1);

    __syncthreads();

    // stage 3. propogate (element-wise add) to all
    if (seg_id)
    {
        r_scan += s_scan_shfl[seg_id];
    }
    r_scan_halfwidth += s_scan_shfl[seg_id + seg_num];

    s_scan[local_id] = r_scan;
    s_scan[local_id + blockDim.x] = r_scan_halfwidth;
    if (!local_id)
        s_scan[2 * blockDim.x] = s_scan_shfl[2 * seg_num];

    return;
}
*/

/*  

 --------------- commenting this to try combinning structure and computation stage ------------------

// Trying out ESC algo for our
template <int BLOCK_THREADS>
__global__ void sgpu_SpGEMM_mid2(const int IA[], const int JA[],
    const QValue A[],const int IB[], const int JB[],const QValue B[],
    const int drowIds[], const int gcount,
    const int m, const int n, const int* IC, 
    int JC[], QValue C[]) {

  __shared__ int   s_key[2 * BLOCK_THREADS];
  __shared__ short s_scan[2 * BLOCK_THREADS + 1];
  __shared__ QValue s_val[2 * BLOCK_THREADS];

  //#if __CUDA_ARCH__ >= 300
//	volatile __shared__ int s_scan_shfl[2 * c_scansize / 32 +1];
 // #else
//	volatile __shared__ int *s_scan_shfl;
 // #endif

  const int local_id = threadIdx.x;
  const int group_id = blockIdx.x;
  const int local_size = blockDim.x;
  const int local_id_halfwidth = local_id + local_size;
  const int width = local_size * 2;

  int colID_A;
  int start_row_B;
  int end_row_B;
  int strideB;
  QValue val;
  int invalid_width;

  for(int q = group_id; q < gcount; q += gridDim.x) {
    int rowId = drowIds[q];
    //s_key[local_id] = INT_MAX;
    //s_key[local_id + BLOCK_THREADS] = INT_MAX;
    int local_counter = 0;
    const int ICi = IC[rowId];
    int *iJC = JC + ICi;
    float *iC = C + ICi;
    for(int ap = IA[rowId]; ap < IA[rowId + 1]; ap++) {
        colID_A = JA[ap];
	val = A[ap];  
	start_row_B = IB[colID_A];      
        end_row_B = IB[colID_A + 1];
	
        strideB = end_row_B - start_row_B;

        if(local_id < strideB) {
             s_key[local_counter + local_id] = JB[start_row_B + local_id];
	     s_val[local_counter + local_id] = B[start_row_B + local_id] * val;
        }

        if(local_id_halfwidth < strideB) {
             s_key[local_counter + local_id_halfwidth] = JB[start_row_B + local_id_halfwidth];
	     s_val[local_counter + local_id_halfwidth] = B[start_row_B + local_id_halfwidth] * val;
        }
        local_counter += strideB;
    }
    __syncthreads(); 
    invalid_width = width - local_counter;
    if (local_id < invalid_width) {
	s_key[local_counter + local_id] = n;
    }
    __syncthreads();
    oddeven(s_key, s_val, width);
    __syncthreads();
    bool duplicate = 1;
    bool duplicate_halfwidth = 1;
    if (local_id < local_counter && local_id > 0) {
	    duplicate = (s_key[local_id] != s_key[local_id-1]);
    }
    if (local_id_halfwidth < local_counter) {
        duplicate_halfwidth = (s_key[local_id_halfwidth] != s_key[local_id_halfwidth - 1]);
    }
    s_scan[local_id] = duplicate;
    s_scan[local_id_halfwidth] = duplicate_halfwidth;
    __syncthreads();
    
 //   for(int offset = blockDim.x; offset > 0; offset >>= 1) {
   //     if(local_id < offset) {
  //         s_scan[local_id] += s_scan[local_id + offset];
     //   }
  //  __syncthreads();
 //   if(threadIdx.x == 0) IC[rowId] = s_scan[0] + 1;
    switch(local_size)
    {
        case 64:
              scan_128(s_scan);
              break;
        case 128:
              scan_256(s_scan);
              break;
        case 256:
              scan_512(s_scan);
              break;
    }
    __syncthreads();
    int   move_pointer;
    short final_position, final_position_halfwidth;
    int   final_key,final_key_halfwidth;
    QValue final_value,final_value_halfwidth;
    if (local_id < local_counter && duplicate == 1)
    {
        final_position = s_scan[local_id];
        final_key = s_key[local_id];
        final_value = s_val[local_id];
        move_pointer = local_id + 1;
        while (s_scan[move_pointer] == s_scan[move_pointer + 1])
        {
            final_value += s_val[move_pointer];
            move_pointer++;
        }
    }
    if (local_id_halfwidth < local_counter && duplicate_halfwidth == 1)
    {
        final_position_halfwidth = s_scan[local_id_halfwidth];
        final_key_halfwidth = s_key[local_id_halfwidth];
        final_value_halfwidth = s_val[local_id_halfwidth];
        move_pointer = local_id_halfwidth + 1;

        while (s_scan[move_pointer] == s_scan[move_pointer + 1] && move_pointer < 2 * local_size)
        {
            final_value_halfwidth += s_val[move_pointer];
            move_pointer++;
	}
    }
    __syncthreads();
    // write final_positions and final_values to s_key and s_val
    if (local_id < local_counter && duplicate == 1)
    {
        s_key[final_position] = final_key;
        s_val[final_position] = final_value;
	//iJC[final_position] = final_key;
	//iC[final_position] = final_value;
    }
    if (local_id_halfwidth < local_counter && duplicate_halfwidth == 1)
    {
        s_key[final_position_halfwidth] = final_key_halfwidth;
        s_val[final_position_halfwidth] = final_value_halfwidth;
        //iJC[final_position_halfwidth] = final_key_halfwidth;
	//iC[final_position_halfwidth] = final_value_halfwidth;
    }
    // writing our results to global memory
    __syncthreads();

    local_counter = s_scan[width] - invalid_width;
    if(local_id < local_counter) {
	iJC[local_id] = s_key[local_id];
	iC[local_id] = s_val[local_id];
    }
    if(local_id_halfwidth < local_counter) {
	iJC[local_id_halfwidth] = s_key[local_id_halfwidth];
	iC[local_id_halfwidth] = s_val[local_id_halfwidth];
    }
  }
  }

*/

// My mid bin

template <int BLOCK_THREADS>
__global__ void sgpu_SpGEMM_mix_mid(const int IA[], const int JA[],
    const QValue A[],const int IB[], const int JB[],const QValue B[],
    const int drowIds[], const int gcount,
    const int m, const int n, int* IC, 
    int JC[], QValue C[]) {

  __shared__ int   s_key[2 * BLOCK_THREADS];
  __shared__ short s_scan[2 * BLOCK_THREADS + 1];
  __shared__ QValue s_val[2 * BLOCK_THREADS];
/*
   #if __CUDA_ARCH__ >= 300
	volatile __shared__ int s_scan_shfl[2 * c_scansize / 32 +1];
   #else
 	volatile __shared__ int *s_scan_shfl;
   #endif
*/
  const int local_id = threadIdx.x;
  const int group_id = blockIdx.x;
  const int local_size = blockDim.x;
  const int local_id_halfwidth = local_id + local_size;
  const int width = local_size * 2;

  int colID_A;
  int start_row_B;
  int end_row_B;
  int strideB;
  QValue val;
  int invalid_width;

  for(int q = group_id; q < gcount; q += gridDim.x) {
    int rowId = drowIds[q];
    //s_key[local_id] = INT_MAX;
    //s_key[local_id + BLOCK_THREADS] = INT_MAX;
    int local_counter = 0;
    const int ICi = q * width;
    int *iJC = JC + ICi;
    QValue *iC = C + ICi;
    for(int ap = IA[rowId]; ap < IA[rowId + 1]; ap++) {
        colID_A = JA[ap];
	val = A[ap];  
	start_row_B = IB[colID_A];      
        end_row_B = IB[colID_A + 1];
	
        strideB = end_row_B - start_row_B;

        if(local_id < strideB) {
             s_key[local_counter + local_id] = JB[start_row_B + local_id];
	     s_val[local_counter + local_id] = B[start_row_B + local_id] * val;
        }

        if(local_id_halfwidth < strideB) {
             s_key[local_counter + local_id_halfwidth] = JB[start_row_B + local_id_halfwidth];
	     s_val[local_counter + local_id_halfwidth] = B[start_row_B + local_id_halfwidth] * val;
        }
        local_counter += strideB;
    }
    __syncthreads(); 
    invalid_width = width - local_counter;
    if (local_id < invalid_width) {
	s_key[local_counter + local_id] = n;
    }
    __syncthreads();
    oddeven(s_key, s_val, width);
    __syncthreads();
    bool duplicate = 1;
    bool duplicate_halfwidth = 1;
    if (local_id < local_counter && local_id > 0) {
	    duplicate = (s_key[local_id] != s_key[local_id-1]);
    }
    if (local_id_halfwidth < local_counter) {
        duplicate_halfwidth = (s_key[local_id_halfwidth] != s_key[local_id_halfwidth - 1]);
    }
    s_scan[local_id] = duplicate;
    s_scan[local_id_halfwidth] = duplicate_halfwidth;
    __syncthreads();
    
   //   for(int offset = blockDim.x; offset > 0; offset >>= 1) {
   //     if(local_id < offset) {
   //           s_scan[local_id] += s_scan[local_id + offset];
   //   }
   //  __syncthreads();
   //   if(threadIdx.x == 0) IC[rowId] = s_scan[0] + 1;
/*
#if __CUDA_ARCH__ >= 300
    scan_double_width_plus1_shfl<short, int>(s_scan, s_scan_shfl, local_id,
                                             duplicate, duplicate_halfwidth, local_size/32);
#else*/
     switch(local_size)
    {
    // case 64 can be removed once its confirmed that its not needed
        case 64:
              scan_128(s_scan);
              break;
        case 128:
              scan_256(s_scan);
              break;
        case 256:
              scan_512(s_scan);
              break;
    }/*
#endif*/

    __syncthreads();
    int   move_pointer;
    short final_position, final_position_halfwidth;
    int   final_key,final_key_halfwidth;
    QValue final_value,final_value_halfwidth;
    if (local_id < local_counter && duplicate == 1)
    {
        final_position = s_scan[local_id];
        final_key = s_key[local_id];
        final_value = s_val[local_id];
        move_pointer = local_id + 1;
        while (s_scan[move_pointer] == s_scan[move_pointer + 1])
        {
            final_value += s_val[move_pointer];
            move_pointer++;
        }
    }
    if (local_id_halfwidth < local_counter && duplicate_halfwidth == 1)
    {
        final_position_halfwidth = s_scan[local_id_halfwidth];
        final_key_halfwidth = s_key[local_id_halfwidth];
        final_value_halfwidth = s_val[local_id_halfwidth];
        move_pointer = local_id_halfwidth + 1;

        while (s_scan[move_pointer] == s_scan[move_pointer + 1] && move_pointer < 2 * local_size)
        {
            final_value_halfwidth += s_val[move_pointer];
            move_pointer++;
	} 
    } 
    __syncthreads();
    // write final_positions and final_values to s_key and s_val
    if (local_id < local_counter && duplicate == 1)
    {
        s_key[final_position] = final_key;
        s_val[final_position] = final_value;
	//iJC[final_position] = final_key;
	//iC[final_position] = final_value;
    }
    if (local_id_halfwidth < local_counter && duplicate_halfwidth == 1)
    {
        s_key[final_position_halfwidth] = final_key_halfwidth;
        s_val[final_position_halfwidth] = final_value_halfwidth;
        //iJC[final_position_halfwidth] = final_key_halfwidth;
	//iC[final_position_halfwidth] = final_value_halfwidth;
    }
    // writing our results to global memory
    __syncthreads();

    //local_counter represents the total number of nnz in the row
    local_counter = s_scan[width] - invalid_width;
    if(local_id == 0) {
	IC[rowId] = local_counter;
    }
    if(local_id < local_counter) {
	iJC[local_id] = s_key[local_id];
	iC[local_id] = s_val[local_id];
    }
    if(local_id_halfwidth < local_counter) {
	iJC[local_id_halfwidth] = s_key[local_id_halfwidth];
	iC[local_id_halfwidth] = s_val[local_id_halfwidth];
    }  
  }
  
//   printf("exiting from kernel");
  }


template <int BLOCK_THREADS,int MUL>
__global__ void sgpu_SpGEMM_copy_mid(
    const int drowIds[], const int gcount,
    const int m, const int n, const int* IC, 
    int JC[], QValue C[], int tempJC[], QValue tempC[]) {

    const int group_id = blockIdx.x;
    const int block_size = blockDim.x;
    const int width = block_size * MUL;
    const int local_id = threadIdx.x;

    for(int q = group_id; q < gcount; q += gridDim.x) {
	int rowId = drowIds[q];
	const int distIC = IC[rowId];
        const int rowSize = IC[rowId + 1] - distIC;
	int tempCopyPos = q * width;
        int *iJC = JC + distIC;
	QValue *iC = C + distIC;
	for(int i = local_id; i < rowSize; i+=block_size) {
	    iJC[i] = tempJC[tempCopyPos + i];
	    iC[i] = tempC[tempCopyPos + i];
	} 
    }
}

template <int BLOCK_THREADS>
__global__ void sgpu_SpGEMM_copy_mid_11(
    const int drowIds[], const int gcount,
    const int m, const int n, const int* IC, 
    int JC[], QValue C[], int tempJC[], QValue tempC[]) {

    const int group_id = blockIdx.x;
    const int block_size = blockDim.x;
    const int width = block_size * 4;
    const int local_id = threadIdx.x;

    for(int q = group_id; q < gcount; q += gridDim.x) {
	int rowId = drowIds[q];
	const int distIC = IC[rowId];
        const int rowSize = IC[rowId + 1] - distIC;
	int tempCopyPos = q * width;
        int *iJC = JC + distIC;
	QValue *iC = C + distIC;
	for(int i = local_id; i < rowSize; i+=block_size) {
	    iJC[i] = tempJC[tempCopyPos + i];
	    iC[i] = tempC[tempCopyPos + i];
	} 
    }
}

template <int BLOCK_THREADS>
__global__ void sgpu_SpGEMM_mix_11(const int IA[], const int JA[],
    const QValue A[],const int IB[], const int JB[],const QValue B[],
    const int drowIds[], const int gcount,
    const int m, const int n, int* IC, 
    int JC[], QValue C[], int *xbs) {

  __shared__ int as[BLOCK_THREADS];
  __shared__ QValue Aaps[BLOCK_THREADS];
  __shared__ int count;

  const int local_id = threadIdx.x;
  const int group_id = blockIdx.x;
  const int local_size = blockDim.x;
  const int width = local_size * 4;

  if(local_id == 0) {
	count = 0;
  }
  int *xb = xbs + group_id * n;

  __syncthreads();

  for(int q = group_id; q < gcount; q += gridDim.x) {
    int rowId = drowIds[q];
    const int ICi = q * width;
    int *iJC = JC + ICi;
    QValue *iC = C + ICi;
    int end_Row = IA[rowId + 1];
    for(int ap = IA[rowId] + local_id; __syncthreads_or(ap < end_Row); ap+= local_size) {
	int predicate = (ap < end_Row);
	int a = predicate ? JA[ap] : -1;
 	QValue Aap = predicate ? A[ap] : 0.0;
	as[local_id] = a;
	Aaps[local_id] = Aap;
	unsigned total = min(end_Row + local_id - ap, local_size);
        __syncthreads();
	for (int ap = 0; ap < total; ++ap) {
        int a = as[ap];
        QValue Aap = Aaps[ap];
        for (int bp = IB[a] + local_id; bp < IB[a + 1]; bp += local_size) {
          int b = JB[bp];
          int xbB = xb[b];
          if (xbB == -1) {
            int pos = atomicAdd(&count, 1);
            iJC[pos] = b;
            iC[pos] = Aap * B[bp];
            xb[b] = pos;
          } else {
            iC[xbB] += Aap * B[bp];
          }
        }
        __syncthreads();
      }
    }
    for (int cp = threadIdx.x; cp < count; cp += blockDim.x) {
      int c = iJC[cp];
      xb[c] = -1;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
      IC[rowId] = count;
      count = 0;
    } 
  }
  }

template <int BLOCK_THREADS>
__global__ void sgpu_SpGEMM_mix_12(const int IA[], const int JA[],
    const QValue A[],const int IB[], const int JB[],const QValue B[],
    const int drowIds[], const int gcount,
    const int m, const int n, int* IC, 
    int JC[], QValue C[], int *xbs) {

  __shared__ int as[BLOCK_THREADS];
  __shared__ QValue Aaps[BLOCK_THREADS];
  __shared__ int count;

  const int local_id = threadIdx.x;
  const int group_id = blockIdx.x;
  const int local_size = blockDim.x;
  const int width = local_size * 8;

  if(local_id == 0) {
	count = 0;
  }
  int *xb = xbs + group_id * n;

  __syncthreads();

  for(int q = group_id; q < gcount; q += gridDim.x) {
    int rowId = drowIds[q];
    const int ICi = q * width;
    int *iJC = JC + ICi;
    QValue *iC = C + ICi;
    int end_Row = IA[rowId + 1];
    for(int ap = IA[rowId] + local_id; __syncthreads_or(ap < end_Row); ap+= local_size) {
	int predicate = (ap < end_Row);
	int a = predicate ? JA[ap] : -1;
 	QValue Aap = predicate ? A[ap] : 0.0;
	as[local_id] = a;
	Aaps[local_id] = Aap;
	unsigned total = min(end_Row + local_id - ap, local_size);
        __syncthreads();
	for (int ap = 0; ap < total; ++ap) {
        int a = as[ap];
        QValue Aap = Aaps[ap];
        for (int bp = IB[a] + local_id; bp < IB[a + 1]; bp += local_size) {
          int b = JB[bp];
          int xbB = xb[b];
          if (xbB == -1) {
            int pos = atomicAdd(&count, 1);
            iJC[pos] = b;
            iC[pos] = Aap * B[bp];
            xb[b] = pos;
          } else {
            iC[xbB] += Aap * B[bp];
          }
        }
        __syncthreads();
      }
    }
    for (int cp = threadIdx.x; cp < count; cp += blockDim.x) {
      int c = iJC[cp];
      xb[c] = -1;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
      IC[rowId] = count;
      count = 0;
    } 
  }
  }
